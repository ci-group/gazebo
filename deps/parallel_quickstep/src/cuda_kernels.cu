#include "hip/hip_runtime.h"
#include <cuda_utils.h>
#include <parallel_common.h>
#include <parallel_utils.h>
#include <parallel_reduce.h>

#include "cuda_kernels.cuh"
#include "cuda_utils.cuh"
#include <math.h>

#include "parallel_kernels.h"
#include "parallel_kernels_nontemplate.h"

namespace parallel_ode
{

using ::parallel_utils::computeGridSize;

__device__ __constant__ int4 B_C_BSTRIDE_CSTRIDE;

void cudaPGSLoadConstants( int numConstraints,
                           int constraintStride,
                           int constraintVecStride,
                           int numBodies,
                           int bodyStride,
                           int bodyVecStride )
{
  int4 b_c_bstride_cstride = make_int4( numBodies, numConstraints, bodyStride, constraintStride );
  CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(B_C_BSTRIDE_CSTRIDE), &b_c_bstride_cstride, sizeof(int4)) );
}

template <typename T>
void cudaPGSReduce( typename vec4<T>::Type *fc0,
                    typename vec4<T>::Type *fc1,
                    typename vec4<T>::Type *fc0_reduction,
                    typename vec4<T>::Type *fc1_reduction,
                    ReduceStrategy* reduceStrategy )
{
  typedef typename vec4<T>::Type Vec4T;

  int numBlocks, numThreads;

  const int bodySize = reduceStrategy->getBodySize( );
  const int bodyReductionSize = reduceStrategy->getBodySizeWithReduction( );
  const int bodyStride = reduceStrategy->getBodyStride( );
  const int bodyOffsetStride = reduceStrategy->getBodyOffsetStride( );

  int smemSize = 0;

  switch( reduceStrategy->getType( ) )
  {
    case ReduceTypes::REDUCE_SEQUENTIAL:
    {
      smemSize = (bodyStride<=32)?4*bodyStride*sizeof(Vec4T):2*bodyStride*sizeof(Vec4T);

      switch( bodyStride )
      {
        case 128:
          cudaReduceSequentialT<T, 128><<< bodySize, bodyStride, smemSize >>>( fc0, fc1, fc0_reduction, fc1_reduction, bodyReductionSize ); break;
        case 64:
          cudaReduceSequentialT<T, 64><<< bodySize, bodyStride, smemSize >>>( fc0, fc1, fc0_reduction, fc1_reduction, bodyReductionSize ); break;
        case 32:
          cudaReduceSequentialT<T, 32><<< bodySize, bodyStride, smemSize >>>( fc0, fc1, fc0_reduction, fc1_reduction, bodyReductionSize ); break;
        case 16:
          cudaReduceSequentialT<T, 16><<< bodySize, bodyStride, smemSize >>>( fc0, fc1, fc0_reduction, fc1_reduction, bodyReductionSize ); break;
        case  8:
          cudaReduceSequentialT<T, 8><<< bodySize, bodyStride, smemSize >>>( fc0, fc1, fc0_reduction, fc1_reduction, bodyReductionSize ); break;
        case  4:
          cudaReduceSequentialT<T, 4><<< bodySize, bodyStride, smemSize >>>( fc0, fc1, fc0_reduction, fc1_reduction, bodyReductionSize ); break;
        case  2:
          cudaReduceSequentialT<T, 2><<< bodySize, bodyStride, smemSize >>>( fc0, fc1, fc0_reduction, fc1_reduction, bodyReductionSize ); break;
        case  1:
          cudaReduceSequentialT<T, 1><<< bodySize, bodyStride, smemSize >>>( fc0, fc1, fc0_reduction, fc1_reduction, bodyReductionSize ); break;
      }
    } break;
    case ReduceTypes::REDUCE_STRIDED:
      computeGridSize( bodySize, 16, numBlocks, numThreads);
      dxExecKernel(numBlocks,
                   numThreads,
                   (cudaReduceStridedT<T>),
                   ( fc0, fc1, fc0_reduction, fc1_reduction, bodyOffsetStride, bodySize, bodyReductionSize ) );
      break;
    case ReduceTypes::REDUCE_COMPACT:
      computeGridSize(bodyReductionSize, ParallelOptions::BSIZE, numBlocks, numThreads);
      for( int treePower = (int)log2( (float)bodyOffsetStride ); treePower > 0; treePower /= 2) {
          dxExecKernel(numBlocks,
                       numThreads,
                       (cudaReduceIterativeCompactT<T>),
                       ( fc0_reduction, fc1_reduction, treePower ) );
      }
      break;
    case ReduceTypes::REDUCE_NONE:    break;

  };

  if( reduceStrategy->clearReduceBuffers( ) ) {
    cudaZeroVector<Vec4T>(fc0_reduction, bodyReductionSize);
    cudaZeroVector<Vec4T>(fc1_reduction, bodyReductionSize);
  }

  CUDA_CHECK_ERROR_BASE();
}

template <typename T>
void cudaZeroVector( T *buffer, int bufferSize )
{
  int numThreads, numBlocks;
  computeGridSize(bufferSize, ParallelOptions::BSIZE, numBlocks, numThreads);

  dxExecKernel( numBlocks,
                numThreads,
                (cudaZeroT<T>),
                ( buffer, bufferSize ) );

  CUDA_CHECK_ERROR_BASE();
}

template <typename T>
void cudaPGSSolve( int4 *bodyIDs,
                   int  *fIDs,
                   typename vec4<T>::Type *j,
                   typename vec4<T>::Type *ij,
                   typename vec4<T>::Type *fc0,
                   typename vec4<T>::Type *fc1,
                   typename vec4<T>::Type *fc0_reduction,
                   typename vec4<T>::Type *fc1_reduction,
                   T* lambda,
                   T* adcfm,
                   T* rhs,
                   T* hilo,
                   int offset, int numConstraints, bool bUseAtomics,
                   int bStride, int cStride)
{
  int numThreads, numBlocks;
  computeGridSize(numConstraints, ParallelOptions::BSIZE, numBlocks, numThreads);

  dxExecKernel(numBlocks,
               numThreads,
               (cudaSORLCPT<T>),
               ( fc0_reduction,
                 fc1_reduction,
                 lambda,
                 bodyIDs,
                 fIDs,
                 j,
                 ij,
                 fc0,
                 fc1,
                 adcfm,
                 rhs,
                 hilo,
                 offset,
                 numConstraints,
                 bStride,
                 cStride
                 ) );

  CUDA_CHECK_ERROR_BASE();
}

template <typename T>
void cudaPGSPreprocess( int4 *bodyIDs,
                        typename vec4<T>::Type *j0,
                        typename vec4<T>::Type *j1,
                        typename vec4<T>::Type *j2,
                        typename vec4<T>::Type *j3,
                        typename vec4<T>::Type *ij0,
                        typename vec4<T>::Type *ij1,
                        typename vec4<T>::Type *ij2,
                        typename vec4<T>::Type *ij3,
                        typename vec4<T>::Type *i0,
                        typename vec4<T>::Type *i1,
                        typename vec4<T>::Type *i2,
                        T* iMass,
                        T* adcfm,
                        T* rhs,
                        T sorParam, T deltaTime, int numConstraints )
{
  int numThreads, numBlocks;
  computeGridSize(numConstraints, ParallelOptions::BSIZE, numBlocks, numThreads);

  dxExecKernel(numBlocks,
               numThreads,
               (cudaComputeInvMJTT<T>),
               (bodyIDs, j0, j1, j2, j3, ij0, i0, i1, i2, iMass, numConstraints, ij1, ij2, ij3 ));

  dxExecKernel(numBlocks,
               numThreads,
               (cudaComputeAdcfmBT<T>),
               ( bodyIDs, j0, j1, j2, j3, ij0, ij1, ij2, ij3, adcfm, rhs, sorParam, numConstraints ));


  CUDA_CHECK_ERROR_BASE();

}

#ifdef CUDA_DOUBLESUPPORT
template void cudaPGSReduce<dReal>( dReal4 *fc0,
                                    dReal4 *fc1,
                                    dReal4 *fc0_reduction,
                                    dReal4 *fc1_reduction,
                                    ReduceStrategy* reduceStrategy );

template void cudaPGSSolve<dReal>( int4 *bodyIDs,
                                   int *fIDs,
                                   dReal4 *j,
                                   dReal4 *ij,
                                   dReal4 *fc0,
                                   dReal4 *fc1,
                                   dReal4 *fc0_reduction,
                                   dReal4 *fc1_reduction,
                                   dReal *lambda,
                                   dReal *adcfm,
                                   dReal *rhs,
                                   dReal *hilo,
                                   int offset, int numConstraints, bool bUseAtomics, int bStride, int cStride );

template void cudaPGSPreprocess<dReal>( int4 *bodyIDs,
                                        dReal4 *j0,
                                        dReal4 *j1,
                                        dReal4 *j2,
                                        dReal4 *j3,
                                        dReal4 *ij0,
                                        dReal4 *ij1,
                                        dReal4 *ij2,
                                        dReal4 *ij3,
                                        dReal4 *ii0,
                                        dReal4 *ii1,
                                        dReal4 *ii2,
                                        dReal *adcfm,
                                        dReal *ad,
                                        dReal *rhs,
                                        dReal sorParam, dReal deltaTime, int numConstraints );

template void cudaZeroVector<dReal4>( dReal4 *buffer, int bufferSize );
#else
template void cudaPGSReduce<float>( float4 *fc0,
                                    float4 *fc1,
                                    float4 *fc0_reduction,
                                    float4 *fc1_reduction,
                                    ReduceStrategy* reduceStrategy );

template void cudaPGSSolve<float>( int4 *bodyIDs,
                                   int *fIDs,
                                   float4 *j,
                                   float4 *ij,
                                   float4 *fc0,
                                   float4 *fc1,
                                   float4 *fc0_reduction,
                                   float4 *fc1_reduction,
                                   float *lambda,
                                   float *adcfm,
                                   float *rhs,
                                   float *hilo,
                                   int offset, int numConstraints, bool bUseAtomics, int bStride, int cStride );

template void cudaPGSPreprocess<float>( int4 *bodyIDs,
                                        float4 *j0,
                                        float4 *j1,
                                        float4 *j2,
                                        float4 *j3,
                                        float4 *ij0,
                                        float4 *ij1,
                                        float4 *ij2,
                                        float4 *ij3,
                                        float4 *ii0,
                                        float4 *ii1,
                                        float4 *ii2,
                                        float *adcfm,
                                        float *ad,
                                        float *rhs,
                                        float sorParam, float deltaTime, int numConstraints );

template void cudaZeroVector<float4>( float4 *buffer, int bufferSize );
#endif

}
